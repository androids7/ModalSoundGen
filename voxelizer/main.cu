#include "hip/hip_runtime.h"
#include "includes/CompFab.h"
#include "math.h"
#include "hiprand.h"
#include "hiprand/hiprand_kernel.h"
#include "includes/cuda_math.h"

#include <iostream>
#include <string>
#include <sstream>
#include "stdio.h"
#include <vector>

#define RANDOM_SEEDS 1000
#define EPSILONF 0.00000000001
#define E_PI 3.1415926535897932384626433832795028841971693993751058209749445923078164062

// check cuda calls for errors
#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }

#define X 0
#define Y 1
#define Z 2

#define CROSS(dest,v1,v2) \
          dest[0]=v1[1]*v2[2]-v1[2]*v2[1]; \
          dest[1]=v1[2]*v2[0]-v1[0]*v2[2]; \
          dest[2]=v1[0]*v2[1]-v1[1]*v2[0];

#define DOT(v1,v2) (v1[0]*v2[0]+v1[1]*v2[1]+v1[2]*v2[2])

#define SUB(dest,v1,v2) \
          dest[0]=v1[0]-v2[0]; \
          dest[1]=v1[1]-v2[1]; \
          dest[2]=v1[2]-v2[2];

#define FINDMINMAX(x0,x1,x2,min,max) \
  min = max = x0;   \
  if(x1<min) min=x1;\
  if(x1>max) max=x1;\
  if(x2<min) min=x2;\
  if(x2>max) max=x2;

__device__ int planeBoxOverlap(float normal[3],float d, float maxbox[3])
{
  int q;
  float vmin[3],vmax[3];
  for(q=X;q<=Z;q++)
  {
    if(normal[q]>0.0f)
    {
      vmin[q]=-maxbox[q];
      vmax[q]=maxbox[q];
    }
    else
    {
      vmin[q]=maxbox[q];
      vmax[q]=-maxbox[q];
    }
  }
  if(DOT(normal,vmin)+d>0.0f) return 0;
  if(DOT(normal,vmax)+d>=0.0f) return 1;

  return 0;
}


/*======================== X-tests ========================*/
#define AXISTEST_X01(a, b, fa, fb)             \
    p0 = a*v0[Y] - b*v0[Z];                    \
    p2 = a*v2[Y] - b*v2[Z];                    \
        if(p0<p2) {min=p0; max=p2;} else {min=p2; max=p0;} \
    rad = fa * boxhalfsize[Y] + fb * boxhalfsize[Z];   \
    if(min>rad || max<-rad) return 0;

#define AXISTEST_X2(a, b, fa, fb)              \
    p0 = a*v0[Y] - b*v0[Z];                    \
    p1 = a*v1[Y] - b*v1[Z];                    \
        if(p0<p1) {min=p0; max=p1;} else {min=p1; max=p0;} \
    rad = fa * boxhalfsize[Y] + fb * boxhalfsize[Z];   \
    if(min>rad || max<-rad) return 0;

/*======================== Y-tests ========================*/
#define AXISTEST_Y02(a, b, fa, fb)             \
    p0 = -a*v0[X] + b*v0[Z];                   \
    p2 = -a*v2[X] + b*v2[Z];                       \
        if(p0<p2) {min=p0; max=p2;} else {min=p2; max=p0;} \
    rad = fa * boxhalfsize[X] + fb * boxhalfsize[Z];   \
    if(min>rad || max<-rad) return 0;

#define AXISTEST_Y1(a, b, fa, fb)              \
    p0 = -a*v0[X] + b*v0[Z];                   \
    p1 = -a*v1[X] + b*v1[Z];                       \
        if(p0<p1) {min=p0; max=p1;} else {min=p1; max=p0;} \
    rad = fa * boxhalfsize[X] + fb * boxhalfsize[Z];   \
    if(min>rad || max<-rad) return 0;

/*======================== Z-tests ========================*/

#define AXISTEST_Z12(a, b, fa, fb)             \
    p1 = a*v1[X] - b*v1[Y];                    \
    p2 = a*v2[X] - b*v2[Y];                    \
        if(p2<p1) {min=p2; max=p1;} else {min=p1; max=p2;} \
    rad = fa * boxhalfsize[X] + fb * boxhalfsize[Y];   \
    if(min>rad || max<-rad) return 0;

#define AXISTEST_Z0(a, b, fa, fb)              \
    p0 = a*v0[X] - b*v0[Y];                \
    p1 = a*v1[X] - b*v1[Y];                    \
        if(p0<p1) {min=p0; max=p1;} else {min=p1; max=p0;} \
    rad = fa * boxhalfsize[X] + fb * boxhalfsize[Y];   \
    if(min>rad || max<-rad) return 0;

__device__ int triBoxOverlap(float boxcenter[3],float boxhalfsize[3],float triverts[3][3])
{

  /*    use separating axis theorem to test overlap between triangle and box */
  /*    need to test for overlap in these directions: */
  /*    1) the {x,y,z}-directions (actually, since we use the AABB of the triangle */
  /*       we do not even need to test these) */
  /*    2) normal of the triangle */
  /*    3) crossproduct(edge from tri, {x,y,z}-directin) */
  /*       this gives 3x3=9 more tests */
   float v0[3],v1[3],v2[3];
   float min,max,d,p0,p1,p2,rad,fex,fey,fez;
   float normal[3],e0[3],e1[3],e2[3];

   /* This is the fastest branch on Sun */
   /* move everything so that the boxcenter is in (0,0,0) */
   SUB(v0,triverts[0],boxcenter);
   SUB(v1,triverts[1],boxcenter);
   SUB(v2,triverts[2],boxcenter);

   /* compute triangle edges */
   SUB(e0,v1,v0);      /* tri edge 0 */
   SUB(e1,v2,v1);      /* tri edge 1 */
   SUB(e2,v0,v2);      /* tri edge 2 */

   /* Bullet 3:  */
   /*  test the 9 tests first (this was faster) */
   fex = fabs(e0[X]);
   fey = fabs(e0[Y]);
   fez = fabs(e0[Z]);
   AXISTEST_X01(e0[Z], e0[Y], fez, fey);
   AXISTEST_Y02(e0[Z], e0[X], fez, fex);
   AXISTEST_Z12(e0[Y], e0[X], fey, fex);

   fex = fabs(e1[X]);
   fey = fabs(e1[Y]);
   fez = fabs(e1[Z]);
   AXISTEST_X01(e1[Z], e1[Y], fez, fey);
   AXISTEST_Y02(e1[Z], e1[X], fez, fex);
   AXISTEST_Z0(e1[Y], e1[X], fey, fex);

   fex = fabs(e2[X]);
   fey = fabs(e2[Y]);
   fez = fabs(e2[Z]);
   AXISTEST_X2(e2[Z], e2[Y], fez, fey);
   AXISTEST_Y1(e2[Z], e2[X], fez, fex);
   AXISTEST_Z12(e2[Y], e2[X], fey, fex);

   /* Bullet 1: */
   /*  first test overlap in the {x,y,z}-directions */
   /*  find min, max of the triangle each direction, and test for overlap in */
   /*  that direction -- this is equivalent to testing a minimal AABB around */
   /*  the triangle against the AABB */

   /* test in X-direction */
   FINDMINMAX(v0[X],v1[X],v2[X],min,max);
   if(min>boxhalfsize[X] || max<-boxhalfsize[X]) return 0;

   /* test in Y-direction */
   FINDMINMAX(v0[Y],v1[Y],v2[Y],min,max);
   if(min>boxhalfsize[Y] || max<-boxhalfsize[Y]) return 0;

   /* test in Z-direction */
   FINDMINMAX(v0[Z],v1[Z],v2[Z],min,max);
   if(min>boxhalfsize[Z] || max<-boxhalfsize[Z]) return 0;

   /* Bullet 2: */
   /*  test if the box intersects the plane of the triangle */
   /*  compute plane equation of triangle: normal*x+d=0 */
   CROSS(normal,e0,e1);
   d=-DOT(normal,v0);  /* plane eq: normal.x+d=0 */
   if(!planeBoxOverlap(normal,d,boxhalfsize)) return 0;

   return 1;   /* box and triangle overlaps */
}

inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

// generates a random float between 0 and 1
__device__ float generate( hiprandState* globalState , int ind) 
{
    hiprandState localState = globalState[ind];
    float RANDOM = hiprand_uniform( &localState );
    globalState[ind] = localState; 
    return RANDOM;
}
// set up random seed buffer
__global__ void setup_kernel ( hiprandState * state, unsigned long seed )
{
    int id = threadIdx.x;
    hiprand_init ( seed, id, 0, &state[id] );
} 


__device__ bool inside(unsigned int numIntersections, bool double_thick) {
	// if (double_thick && numIntersections % 2 == 0) return (numIntersections / 2) % 2 == 1;
	if (double_thick) return (numIntersections / 2) % 2 == 1;
	return numIntersections % 2 == 1;
}

// adapted from: https://en.wikipedia.org/wiki/M%C3%B6ller%E2%80%93Trumbore_intersection_algorithm
__device__ bool intersects(CompFab::Triangle &triangle, float3 dir, float3 pos) {
	float3 V1 = {triangle.m_v1.m_x, triangle.m_v1.m_y, triangle.m_v1.m_z};
	float3 V2 = {triangle.m_v2.m_x, triangle.m_v2.m_y, triangle.m_v2.m_z};
	float3 V3 = {triangle.m_v3.m_x, triangle.m_v3.m_y, triangle.m_v3.m_z};

	//Find vectors for two edges sharing V1
	float3 e1 = V2 - V1;
	float3 e2 = V3 - V1;
	
	// //Begin calculating determinant - also used to calculate u parameter
	float3 P = cross(dir, e2);

	//if determinant is near zero, ray lies in plane of triangle
	float det = dot(e1, P);
	
	//NOT CULLING
	if(det > -EPSILONF && det < EPSILONF) return false;
	float inv_det = 1.f / det;

	// calculate distance from V1 to ray origin
	float3 T = pos - V1;
	//Calculate u parameter and test bound
	float u = dot(T, P) * inv_det;
	//The intersection lies outside of the triangle
	if(u < 0.f || u > 1.f) return false;

	//Prepare to test v parameter
	float3 Q = cross(T, e1);
	//Calculate V parameter and test bound
	float v = dot(dir, Q) * inv_det;
	//The intersection lies outside of the triangle
	if(v < 0.f || u + v  > 1.f) return false;

	float t = dot(e2, Q) * inv_det;

	if(t > EPSILONF) { // ray intersection
		return true;
	}

	// No hit, no win
	return false;
}

__device__ float distance( CompFab::Triangle &triangle ,  float3 pos) 
{
	float3 V1 = {triangle.m_v1.m_x, triangle.m_v1.m_y, triangle.m_v1.m_z};
	float3 V2 = {triangle.m_v2.m_x, triangle.m_v2.m_y, triangle.m_v2.m_z};
	float3 V3 = {triangle.m_v3.m_x, triangle.m_v3.m_y, triangle.m_v3.m_z}; 
	float3 mid = (V1+V2+V3)/3.f;
	float3 d = pos - mid;
	return length(d);
	
}


// Decides whether or not each voxel is within the given mesh
__global__ void voxelize_kernel( 
	bool* R,int* I, CompFab::Triangle* triangles, const int numTriangles, 
	const float spacing, const float3 bottom_left,
	const int w, const int h, const int d, bool double_thick)
{
	// find the position of the voxel
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;
	// pick an arbitrary sampling direction
	float3 dir = make_float3(1.0, 0.0, 0.0);

	if ( (xIndex < w) && (yIndex < h) && (zIndex < d) )
	{
		// find linearlized index in final boolean array
		unsigned int index_out = zIndex*(w*h)+yIndex*h + xIndex;
		
		// find world space position of the voxel
		float3 pos = make_float3(bottom_left.x + spacing*xIndex,bottom_left.y + spacing*yIndex,bottom_left.z + spacing*zIndex);

		// check if the voxel is inside of the mesh. 
		// if it is inside, then there should be an odd number of 
		// intersections with the surrounding mesh
		unsigned int intersections = 0;
		bool yes = 0;
		for (int i = 0; i < numTriangles; ++i){
			if (intersects(triangles[i], dir, pos)){
				intersections += 1;
			}
			float boxcenter[3];
			float boxhalfsize[3];
			boxcenter[0] = pos.x;boxcenter[1] = pos.y;boxcenter[2] = pos.z;
			boxhalfsize[0] = boxhalfsize[1] = boxhalfsize[2] = spacing*0.5;
			CompFab::Triangle &triangle = triangles[i];
			float triverts[3][3] = {
			{triangle.m_v1.m_x, triangle.m_v1.m_y, triangle.m_v1.m_z},
			{triangle.m_v2.m_x, triangle.m_v2.m_y, triangle.m_v2.m_z},
			{triangle.m_v3.m_x, triangle.m_v3.m_y, triangle.m_v3.m_z}};
			if (triBoxOverlap(boxcenter,boxhalfsize,triverts) == 1)
				yes = 1;

		}
		// store answer
		R[index_out] = inside(intersections, double_thick) || yes;
///////////////////////////////////////////////////////////////////////////
/*		I[index_out] = -1;
		if (R[index_out]){
			float mndist = distance(triangles[0],pos);
			int index = 0;
			for (int i = 1; i < numTriangles; ++i)
				if (distance(triangles[i],pos) < mndist){
					mndist = distance(triangles[i],pos);
					index = i;
				}

			// store answer
			I[index_out] = index;

		}*/
///////////////////////////////////////////////////////////////////////////
	}
}


// Decides whether or not each voxel is within the given partially un-closed mesh
// checks a variety of directions and picks most common belief
__global__ void voxelize_kernel_open_mesh( 
	// triangles of the mesh being voxelized
	bool* R, int* I, CompFab::Triangle* triangles, const int numTriangles, 
	// information about how large the samples are and where they begin
	const float spacing, const float3 bottom_left,
	// number of voxels
	const int w, const int h, const int d, 
	// sampling information for multiple intersection rays
	const int samples, hiprandState* globalState, bool double_thick
	)
{
	// find the position of the voxel
	unsigned int xIndex = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned int yIndex = blockDim.y * blockIdx.y + threadIdx.y;
	unsigned int zIndex = blockDim.z * blockIdx.z + threadIdx.z;

	if ( (xIndex < w) && (yIndex < h) && (zIndex < d) )
	{
		// find linearlized index in final boolean array
		unsigned int index_out = zIndex*(w*h)+yIndex*h + xIndex;
		// find world space position of the voxel
		float3 pos = make_float3(bottom_left.x + spacing*xIndex,bottom_left.y + spacing*yIndex,bottom_left.z + spacing*zIndex);
		float3 dir;

		// we will randomly sample 3D space by sending rays in randomized directions
		int votes = 0;
		float theta;
		float z;

		for (int j = 0; j < samples; ++j)
		{
			// compute the random direction. Convert from polar to euclidean to get an even distribution
			theta = generate(globalState, index_out % RANDOM_SEEDS) * 2.f * E_PI;
			z = generate(globalState, index_out % RANDOM_SEEDS) * 2.f - 1.f;

			dir.x = sqrt(1-z*z) * cosf(theta);
			dir.y = sqrt(1-z*z) * sinf(theta);
			dir.z = sqrt(1-z*z) * cosf(theta);

			// check if the voxel is inside of the mesh. 
			// if it is inside, then there should be an odd number of 
			// intersections with the surrounding mesh
			unsigned int intersections = 0;
			for (int i = 0; i < numTriangles; ++i)
				if (intersects(triangles[i], dir, pos)) 
					intersections += 1;
			if (inside(intersections, double_thick)) votes += 1;
		}
		// choose the most popular answer from all of the randomized samples
		R[index_out] = votes > (samples / 2.f);
	}
}

// voxelize the given mesh with the given resolution and dimensions
void kernel_wrapper(int samples, int w, int h, int d, CompFab::VoxelGrid *g_voxelGrid, CompFab::IndexVoxelGrid *g_index_voxelGrid, std::vector<CompFab::Triangle> triangles, bool double_thick)
{
	int blocksInX = (w+8-1)/8;
	int blocksInY = (h+8-1)/8;
	int blocksInZ = (d+8-1)/8;

	dim3 Dg(blocksInX, blocksInY, blocksInZ);
	dim3 Db(8, 8, 8);

	hiprandState* devStates;
	if (samples > 0) {
		// set up random numbers
		dim3 tpb(RANDOM_SEEDS,1,1);
	    hipMalloc ( &devStates, RANDOM_SEEDS*sizeof( hiprandState ) );
	    // setup seeds
	    setup_kernel <<< 1, tpb >>> ( devStates, time(NULL) );
	}
	
	// set up boolean array on the GPU
	bool *gpu_inside_array;
	gpuErrchk( hipMalloc( (void **)&gpu_inside_array, sizeof(bool) * w * h * d ) );
	gpuErrchk( hipMemcpy( gpu_inside_array, g_voxelGrid->m_insideArray, sizeof(bool) * w * h * d, hipMemcpyHostToDevice ) );

//////////////////////////////////////////////////////////////////////////////////////////
	//// set up int array on the GPU
	int *gpu_inside_index_array;
	gpuErrchk( hipMalloc( (void **)&gpu_inside_index_array, sizeof(int) * w * h * d ) );
	gpuErrchk( hipMemcpy( gpu_inside_index_array, g_index_voxelGrid->m_insideArray, sizeof(int) * w * h * d, hipMemcpyHostToDevice ) );
	gpuErrchk( hipMemcpy( gpu_inside_array, g_voxelGrid->m_insideArray, sizeof(bool) * w * h * d, hipMemcpyHostToDevice ) );
////////////////////////////////////////////////////////////////////////////////////////////
	// set up triangle array on the GPU
	CompFab::Triangle* triangle_array = &triangles[0];
	CompFab::Triangle* gpu_triangle_array;
	gpuErrchk( hipMalloc( (void **)&gpu_triangle_array, sizeof(CompFab::Triangle) * triangles.size() ) );
	gpuErrchk( hipMemcpy( gpu_triangle_array, triangle_array, sizeof(CompFab::Triangle) * triangles.size(), hipMemcpyHostToDevice ) );

	float3 lower_left = make_float3(g_voxelGrid->m_lowerLeft.m_x, g_voxelGrid->m_lowerLeft.m_y, g_voxelGrid->m_lowerLeft.m_z);
	//printf("spacing:%lf\n",(float) g_voxelGrid->m_spacing);
	if (samples > 0) {
		voxelize_kernel_open_mesh<<<Dg, Db>>>(gpu_inside_array, gpu_inside_index_array, gpu_triangle_array, triangles.size(), (float) g_voxelGrid->m_spacing, lower_left, w, h, d, samples, devStates, double_thick);
	} else {
		voxelize_kernel<<<Dg, Db>>>(gpu_inside_array, gpu_inside_index_array, gpu_triangle_array, triangles.size(), (float) g_voxelGrid->m_spacing, lower_left, w, h, d, double_thick);
	}

	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipDeviceSynchronize() );

	gpuErrchk( hipMemcpy( g_voxelGrid->m_insideArray, gpu_inside_array, sizeof(bool) * w * h * d, hipMemcpyDeviceToHost ) );
//////////////////////////////////////////////////////////////////////////////////////////////////////

	gpuErrchk( hipMemcpy( g_index_voxelGrid->m_insideArray, gpu_inside_index_array, sizeof(int) * w * h * d, hipMemcpyDeviceToHost ) );
	gpuErrchk( hipFree(gpu_inside_index_array) );
//////////////////////////////////////////////////////////////////////////////////////////////
	gpuErrchk( hipFree(gpu_inside_array) );
	gpuErrchk( hipFree(gpu_triangle_array) );
}
